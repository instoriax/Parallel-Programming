#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float x0, float y0, float dx, float dy, int count , int resX, int resY, int* output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=resX)
        return;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(j>=resY)
        return;
    float c_re = x0 + i * dx;
    float c_im = y0 + j * dy;
    float z_re = c_re, z_im = c_im;
    int it;
    for (it = 0; it < count; ++it)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
        break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    output[j*resX + i]=it;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int *output;
    hipMalloc(&output, resX*resY*sizeof(int));
    int *host=(int*)malloc(resX*resY*sizeof(int));
    dim3 threadsPerBlock(32, 32);
    int x=resX%32? (resX/32)+1 : (resX/32);
    int y=resY%32? (resY/32)+1 : (resY/32);
    dim3 numBlocks(x,y);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, maxIterations, resX, resY, output);
    hipMemcpy(host, output, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img, host, resX*resY*sizeof(int));
    hipFree(output);
    free(host);
}
